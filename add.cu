#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>

#define _CHUNK_SIZE 1024 * 1024 * 50

class FH {
private:
    std::ifstream _file;
    const uint _chunksize = _CHUNK_SIZE;
public:
    FH(const std::string &filename) {
        _file = std::ifstream(filename);
        char* buffer = new char[_CHUNK_SIZE];
        _file.rdbuf()->pubsetbuf(buffer, _CHUNK_SIZE);
    }

    bool eof() {
        return _file.eof();
    }
    std::vector<float> parse_line_of_floats(const std::string& line) {
        std::vector<float> numbers;
        const char* str = line.c_str();
        char* end;

        while (true) {
            float num = std::strtof(str, &end);
            if ( end == str) break;
            numbers.push_back(num);
            str = end;
        }

        return numbers;
    }
    static size_t total_vector_size(std::vector<std::vector<float> > data) {
        size_t total_size = 0;
        for (std::vector<float> _d : data) {
            total_size += _d.size();
        }
        return total_size;
    }

    std::vector<float> readDataFromFile() {
        std::vector<std::vector<float> > data;
        std::string line;

        size_t total_size = 0;

        while (total_size < _chunksize) {
            if (!(std::getline(_file, line))) break;

            std::vector<float> numbers = parse_line_of_floats(line);

            if (data.size() == 0) {
                data.resize(numbers.size());
            }

            for (int i = 0; i < numbers.size(); i++) {
                data[i].push_back(numbers[i]);
            }

            total_size = total_vector_size(data);
        }

        return data[0];
    }
};

__global__ void addArrays(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

__global__ void complexPhysicsCalculation(const float *A, const float *B, float* C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        // Hypothetical physics-based calculation
        float temp = A[i] * expf(-B[i] / A[i]);
        float result = sinf(A[i]) * cosf(B[i]) + temp;

        // Store the result
        C [i]= result;
    }
}

// Main function
int main(int argc, char *argv[]) {
    // Check for correct argument count
    if (argc != 4) {
        std::cerr << "Usage: " << argv[0] << " <file1> <file2> <output_file>" << std::endl;
        return 1;
    }

    // Read data from files
    FH host_A_file(argv[1]);
    FH host_B_file(argv[2]);

    // Write result to file
    std::ofstream outputFile(argv[3]);

    char* buffer = new char[_CHUNK_SIZE];
    outputFile.rdbuf()->pubsetbuf(buffer, _CHUNK_SIZE);

    // Allocate memory on the GPU
    float *device_A, *device_B, *device_C;
    hipMalloc((void **)&device_A, _CHUNK_SIZE * sizeof(float));
    hipMalloc((void **)&device_B, _CHUNK_SIZE * sizeof(float));
    hipMalloc((void **)&device_C, _CHUNK_SIZE * sizeof(float));

    while (!host_A_file.eof()) {
        std::vector<float> host_A = host_A_file.readDataFromFile();
        std::vector<float> host_B = host_B_file.readDataFromFile();
        int numElements = host_A.size();

        // Copy data from host to device
        hipMemcpy(device_A, host_A.data(), numElements * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(device_B, host_B.data(), numElements * sizeof(float), hipMemcpyHostToDevice);

        // Launch the CUDA Kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
        addArrays<<<blocksPerGrid, threadsPerBlock>>>(device_A, device_B, device_C, numElements);
        // complexPhysicsCalculation<<<blocksPerGrid, threadsPerBlock>>>(device_A, device_B, device_C, numElements);

        // Copy result back to host
        std::vector<float> host_C(numElements);
        hipMemcpy(host_C.data(), device_C, numElements * sizeof(float), hipMemcpyDeviceToHost);

        for (float value : host_C) {
            outputFile << value << "\n";
        }

    }

    // Free device memory
    hipFree(device_A);
    hipFree(device_B);
    hipFree(device_C);

    return 0;
}
