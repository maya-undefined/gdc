#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>

#define _CHUNK_SIZE 1024 * 1024 * 50

class FH {
private:
    std::ifstream _file;
    const uint _chunksize = _CHUNK_SIZE;
public:
    FH(const std::string &filename) {
        _file = std::ifstream(filename);
        char* buffer = new char[_CHUNK_SIZE];
        _file.rdbuf()->pubsetbuf(buffer, _CHUNK_SIZE);
    }

    bool eof() {
        return _file.eof();
    }
    std::vector<float> parse_line_of_floats(const std::string& line) {
        std::vector<float> numbers;
        const char* str = line.c_str();
        char* end = NULL;

        while (true) {
            float num = std::strtof(str, &end);
            // is this safe?

            if ( end == str) break;
            numbers.push_back(num);
            str = end;
        }

        return numbers;
    }
    static size_t total_vector_size(std::vector<std::vector<float> > data) {
        size_t total_size = 0;
        for (std::vector<float> _d : data) {
            total_size += _d.size();
        }
        return total_size;
    }

    std::vector<std::vector<float> > readDataFromFile() {
        std::vector<std::vector<float> > data;
        std::string line;

        size_t total_size = 0;

        while (total_size < _chunksize) {
            if (!(std::getline(_file, line))) break;

            std::vector<float> numbers = parse_line_of_floats(line);

            if (data.size() == 0) {
                data.resize(numbers.size());
            }

            for (int i = 0; i < numbers.size(); i++) {
                data[i].push_back(numbers[i]);
            }

            total_size = total_vector_size(data);
        }

        return data;
    }
};

__global__ void addMultipleArrays(float *A, float *B, float *C, int A_rows, int B_rows, int A_cols, int B_cols) {
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < A_rows) {
        float sum = 0;
        int col, idx;
        for ( col = 0; col < A_cols; ++col) {
            idx = row * A_cols + col;
            sum += A[idx];
        }

        for ( col = 0; col < B_cols; ++col) {
            idx = row * B_cols + col;
            sum += B[idx];
        }

        
        C[row] = sum;
    }
}

__global__ void addArrays(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

__global__ void complexPhysicsCalculation(const float *A, const float *B, float* C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        // Hypothetical physics-based calculation
        float temp = A[i] * expf(-B[i] / A[i]);
        float result = sinf(A[i]) * cosf(B[i]) + temp;

        // Store the result
        C[i]= result;
    }
}

// Main function
int main(int argc, char *argv[]) {
    // Check for correct argument count
    if (argc != 4) {
        std::cerr << "Usage: " << argv[0] << " <file1> <file2> <output_file>" << std::endl;
        return 1;
    }

    // Read data from files
    FH host_A_file(argv[1]);
    FH host_B_file(argv[2]);

    // Write result to file
    std::ofstream outputFile(argv[3]);

    char* buffer = new char[_CHUNK_SIZE];
    outputFile.rdbuf()->pubsetbuf(buffer, _CHUNK_SIZE);

    while (!host_A_file.eof()) {
        std::vector<std::vector<float> > host_A = host_A_file.readDataFromFile();
        std::vector<std::vector<float> > host_B = host_B_file.readDataFromFile();
        int numElements = host_A.size();

        // Allocate memory on the GPU
        float *device_C;
        float *device_A, *device_B;

        hipMalloc((void **)&device_C, _CHUNK_SIZE * sizeof(float));

        std::vector<float> flat_A;
        std::vector<float> flat_B;
        for (const auto& row: host_A) {
            flat_A.insert(flat_A.end(), row.begin(), row.end());
        }

        for (const auto& row: host_B) {
            flat_B.insert(flat_B.end(), row.begin(), row.end());
        }
        
        hipMalloc(&device_A, flat_A.size() * sizeof(float));
        hipMalloc(&device_B, flat_B.size() * sizeof(float));
        hipMemcpy(device_A, flat_A.data(), flat_A.size() * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(device_B, flat_B.data(), flat_B.size() * sizeof(float), hipMemcpyHostToDevice);

        // Launch the CUDA Kernel
        dim3 blockSize(256);
        dim3 gridSize((numElements + blockSize.x - 1) / blockSize.x);
        addMultipleArrays<<<gridSize, blockSize>>>(
                device_A, device_B, device_C, 
                host_A[0].size(), host_B[0].size(),
                host_A.size(), host_B.size() // rows
                ); // columns

        // Copy result back to host
        std::vector<float> host_C(host_A[0].size());
        hipMemcpy(host_C.data(), device_C, host_A.size() * sizeof(float), hipMemcpyDeviceToHost);

        for (float value : host_C) {
            outputFile << value << "\n";
        }

        hipFree(device_A);
        hipFree(device_B);
        hipFree(device_C);
    }



    return 0;
}
