#include "hip/hip_runtime.h"
#include "gdc.h"
#include "FH.h"
#include "verb.h"

// Main function
int main(int argc, char *argv[]) {
    // Check for correct argument count
    if (argc != 5) {
        std::cerr << "Usage: " << argv[0] << " <verb> <file1> <file2> <output_file>" << std::endl;
        std::cerr << "Verb: (add|mul|div|exp)" << std::endl; 
        return 1;
    }

    std::string verb = std::string(argv[1]);
    if (verb != "mul" && verb != "add" && verb != "div" && verb != "exp") {
        std::cerr << "Verb: (add|mul|div|exp)" << std::endl; 
        std::cerr << "Invalid verbs" << std::endl;
    }

    // Write result to file
    std::ofstream outputFile(argv[4]);

    char* buffer = new char[_CHUNK_SIZE];
    outputFile.rdbuf()->pubsetbuf(buffer, _CHUNK_SIZE);
    // Allocate memory on the GPU

    // Read data from files
    FH host_A_file(argv[2]);
    FH host_B_file(argv[3]);

    Add add(argv[2], argv[3], argv[4]);
    add.dispatch();

    // Exp exp(argv[2]);
    // exp.dispatch();

    // size_t A_rows, A_cols, B_rows, B_cols;
    // A_rows = 0; A_cols = 1; B_rows = 0; B_cols = 1;
    // size_t loops = 0;
    // while (!host_A_file.eof()) {
    //     A_rows = host_A_file.row_len();
    //     B_rows = host_B_file.row_len();
    //     // Remember how many rows we read so far

    //     std::vector<float> host_A = host_A_file.read_data_from_file();
    //     std::vector<float> host_B = host_B_file.read_data_from_file();

    //     A_rows = host_A_file.row_len() - A_rows;
    //     B_rows = host_B_file.row_len() - B_rows;
    //     // and now we can calculate how many rows we need to process in this chunk

    //     if (B_cols != host_B_file.col_len()) { B_cols = host_B_file.col_len(); }
    //     if (A_cols != host_A_file.col_len()) { A_cols = host_A_file.col_len(); }

    //     hipMalloc((void **)&device_C, host_A.size() * sizeof(float));
    //     hipMalloc((void **)&device_A, host_A.size() * sizeof(float));
    //     hipMalloc((void **)&device_B, host_B.size() * sizeof(float));
    //     hipMemcpy(device_A, host_A.data(), host_A.size() * sizeof(float), hipMemcpyHostToDevice);
    //     hipMemcpy(device_B, host_B.data(), host_B.size() * sizeof(float), hipMemcpyHostToDevice);

    //     // Launch the CUDA Kernel
    //     dim3 blockSize(256);
    //     dim3 gridSize((A_rows + blockSize.x - 1) / blockSize.x);
    //     if (verb == "add") {
    //         addMultipleArrays<<<gridSize, blockSize>>>(
    //                 device_A, device_B, device_C, 
    //                 A_rows, B_rows, // rows
    //                 A_cols, B_cols // columns
    //                 ); 
    //     } else
    //     if (verb == "mul") {
    //         mulMultipleArrays<<<gridSize, blockSize>>>(
    //             device_A, device_B, device_C, 
    //             A_rows, B_rows, // rows
    //             A_cols, B_cols // columns
    //             ); 
    //     } else
    //     if (verb == "div") {
    //         divArrays<<<gridSize, blockSize>>>(
    //             device_A, device_B, device_C,
    //             A_rows, B_rows
    //         );
    //     } else
    //     if (verb == "exp") {
    //         expArrays<<<gridSize, blockSize>>>(
    //             device_A, device_C,
    //             A_rows
    //         );
    //     }

    //     // Copy result back to host
    //     // we only need to keep track of how many elements since we are using a flat array
    //     size_t ele_to_read = max(A_rows, B_rows);
    //     std::vector<float> host_C(ele_to_read);
    //     hipMemcpy(host_C.data(), device_C, ele_to_read * sizeof(float), hipMemcpyDeviceToHost);

    //     for (float value : host_C) {
    //         outputFile << std::fixed << std::setprecision(6) << value << "\n";
    //     }
    //     loops++;
    //     hipFree(device_A);
    //     hipFree(device_B);
    //     hipFree(device_C);
    // }

    return 0;
}
